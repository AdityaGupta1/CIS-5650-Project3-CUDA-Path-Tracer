#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <glm/gtx/projection.hpp>
#include "utilities.h"
#include "extrautils.hpp"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include "toggles.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;

static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;

static Triangle* dev_tris = NULL;

static BvhNode* dev_bvh_nodes = NULL;
static int* dev_bvh_tri_idx = NULL;

static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

static thrust::device_ptr<PathSegment> dev_thrust_paths = NULL;
static thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections = NULL;

static int numTextures;
static hipTextureObject_t* host_textureObjects = NULL; // array owned by host
static hipTextureObject_t* dev_textureObjects = NULL; // array owned by device
static hipArray_t* host_textureArrayPtrs = NULL; // array owned by host

#if FIRST_BOUNCE_CACHE
static bool fbcNeedsRefresh = true;
static ShadeableIntersection* dev_intersections_fbc = NULL;
static thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections_fbc = NULL;
#endif


void Pathtracer::InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void Pathtracer::init(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_tris, scene->tris.size() * sizeof(Triangle));
	hipMemcpy(dev_tris, scene->tris.data(), scene->tris.size() * sizeof(Triangle), hipMemcpyHostToDevice);

	hipMalloc(&dev_bvh_nodes, scene->bvhNodes.size() * sizeof(BvhNode));
	hipMemcpy(dev_bvh_nodes, scene->bvhNodes.data(), scene->bvhNodes.size() * sizeof(BvhNode), hipMemcpyHostToDevice);
	hipMalloc(&dev_bvh_tri_idx, scene->bvhTriIdx.size() * sizeof(int));
	hipMemcpy(dev_bvh_tri_idx, scene->bvhTriIdx.data(), scene->bvhTriIdx.size() * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	dev_thrust_paths = thrust::device_pointer_cast(dev_paths);
	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	dev_thrust_intersections = thrust::device_pointer_cast(dev_intersections);

	initTextures(scene);

#if FIRST_BOUNCE_CACHE
	hipMalloc(&dev_intersections_fbc, pixelcount * sizeof(ShadeableIntersection));
	dev_thrust_intersections_fbc = thrust::device_pointer_cast(dev_intersections_fbc);
#endif

	checkCUDAError("pathtraceInit");
}

void Pathtracer::initTextures(Scene* scene)
{
	numTextures = scene->textures.size();
	if (numTextures == 0)
	{
		return;
	}

	host_textureObjects = new hipTextureObject_t[numTextures];
	host_textureArrayPtrs = new hipArray_t[numTextures];

	for (int i = 0; i < numTextures; ++i)
	{
		const Texture& texture = scene->textures[i];

		// wasn't working with linear memory so changed to array
		// https://stackoverflow.com/questions/63408787/texture-object-fetching-in-cuda
		auto channelDesc = hipCreateChannelDesc<uchar4>();
		hipMallocArray(&host_textureArrayPtrs[i], &channelDesc, texture.width, texture.height);
		hipMemcpy2DToArray(host_textureArrayPtrs[i], 0, 0, texture.host_dataPtr, texture.width * sizeof(uchar4), texture.width * sizeof(uchar4), texture.height, hipMemcpyHostToDevice);

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = host_textureArrayPtrs[i];

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.readMode = hipReadModeNormalizedFloat;
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.normalizedCoords = 1;

		hipCreateTextureObject(&host_textureObjects[i], &resDesc, &texDesc, NULL);
	}

	hipMalloc((void**)&dev_textureObjects, numTextures * sizeof(hipTextureObject_t));
	hipMemcpy(dev_textureObjects, host_textureObjects, numTextures * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);
}

void Pathtracer::free() {
	hipFree(dev_image);  // no-op if dev_image is null

	hipFree(dev_geoms);
	hipFree(dev_materials);

	hipFree(dev_tris);

	hipFree(dev_bvh_nodes);
	hipFree(dev_bvh_tri_idx);

	hipFree(dev_paths);
	hipFree(dev_intersections);

	freeTextures();

#if FIRST_BOUNCE_CACHE
	hipFree(dev_intersections_fbc);
#endif

	checkCUDAError("pathtraceFree");
}

void Pathtracer::freeTextures()
{
	if (numTextures == 0)
	{
		return;
	}

	hipFree(dev_textureObjects);

	for (int i = 0; i < numTextures; ++i)
	{
		hipDestroyTextureObject(host_textureObjects[i]);
		hipFreeArray(host_textureArrayPtrs[i]);
	}

	delete[] host_textureObjects;
	delete[] host_textureArrayPtrs;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= cam.resolution.x || y >= cam.resolution.y) {
		return;
	}

	int index = x + (y * cam.resolution.x);
	PathSegment& segment = pathSegments[index];

	segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

	thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
	thrust::uniform_real_distribution<float> u01(0, 1);

	glm::vec3 noLensDirection = glm::normalize(cam.view
		- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + u01(rng))
		- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + u01(rng))
	);

	if (cam.lensRadius > 0)
	{
		float z = glm::length(glm::proj(noLensDirection, cam.view));
		glm::vec3 pFocus = cam.position + (noLensDirection * cam.focusDistance / z);
		glm::vec2 pLens = cam.lensRadius * ConcentricSampleDisk(glm::vec2(u01(rng), u01(rng)));

		Ray newRay;
		newRay.origin = cam.position + (pLens.x * cam.right + pLens.y * cam.up);
		newRay.direction = glm::normalize(pFocus - newRay.origin);
		segment.ray = newRay;
	}
	else
	{
		segment.ray = { cam.position, noLensDirection };
	}

	segment.pixelIndex = index;
	segment.bouncesSoFar = 0;
	segment.remainingBounces = traceDepth;
}

__global__ void computeIntersections(
	int depth, 
	int num_paths, 
	PathSegment* pathSegments, 
	Geom* geoms,
	int geoms_size,
	Triangle* tris,
	BvhNode* bvhNodes,
	int* bvhTriIdx,
	ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index >= num_paths)
	{
		return;
	}

	PathSegment pathSegment = pathSegments[path_index];

	float t;
	glm::vec3 intersect_point;
	glm::vec3 normal;
	glm::vec2 uv;
	int triIdx;
	float t_min = FLT_MAX;
	int hit_geom_index = -1;

	glm::vec3 tmp_intersect;
	glm::vec3 tmp_normal;
	glm::vec2 tmp_uv;
	int tmp_triIdx;

	// naive parse through global geoms

	for (int i = 0; i < geoms_size; i++)
	{
		Geom& geom = geoms[i];

		if (geom.type == CUBE)
		{
			t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal);
		}
		else if (geom.type == SPHERE)
		{
			t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal);
		}
		else if (geom.type == MESH)
		{
			t = meshIntersectionTest(geom, tris, bvhNodes, bvhTriIdx, pathSegment.ray, tmp_intersect, tmp_normal, tmp_uv, tmp_triIdx);
		}

		if (t < 0 || t > t_min)
		{
			continue;
		}

		t_min = t;
		hit_geom_index = i;
		intersect_point = tmp_intersect;
		normal = tmp_normal;
		uv = tmp_uv;
		triIdx = tmp_triIdx;
	}

	if (hit_geom_index == -1)
	{
		intersections[path_index].t = -1.0f;
	}
	else
	{
		//The ray hits something
		ShadeableIntersection& isect = intersections[path_index];
		isect.hitGeomIdx = hit_geom_index;
		isect.t = t_min;
		isect.materialId = geoms[hit_geom_index].materialId;
		isect.surfaceNormal = normal;
		isect.uv = uv;
		isect.triIdx = triIdx;
	}
}

struct SegmentProcessingSettings
{
	bool russianRoulette;
};

__device__ void processSegment(
	PathSegment& segment, 
	ShadeableIntersection& intersection,
	Geom* geoms,
	Triangle* tris,
	Material* materials, 
	hipTextureObject_t* textureObjects, 
	int iter, 
	int idx, 
	SegmentProcessingSettings settings)
{
	if (intersection.t <= 0.0f)
	{
		segment.color = glm::vec3(0.0f);
		segment.remainingBounces = 0;
		return;
	}

	thrust::uniform_real_distribution<float> u01(0, 1);
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, segment.bouncesSoFar + 1);

	Material material = materials[intersection.materialId];

	if (material.emission.strength > 0)
	{
		segment.color *= material.emission.color * material.emission.strength;
		segment.remainingBounces = 0;
		return;
	} 
	
	scatterRay(
		segment,
		intersection,
		getPointOnRay(segment.ray, intersection.t),
		geoms,
		tris,
		material,
		textureObjects,
		rng
	);

#if DEBUG_SHOW_NORMALS
		segment.color = (intersection.surfaceNormal + 1.f) / 2.f;
		segment.remainingBounces = 0;
		return;
#endif

	++segment.bouncesSoFar;

	if (--segment.remainingBounces == 0)
	{
		segment.color = glm::vec3(0.0f);
		return;
	}

	if (settings.russianRoulette && segment.bouncesSoFar > 3)
	{
		float q = glm::max(0.05f, 1 - Utils::luminance(segment.color));
		if (u01(rng) < q)
		{
			segment.color = glm::vec3(0.0f);
			segment.remainingBounces = 0;
			return;
		}

		segment.color /= (1 - q);
	}
}

__global__ void shadeMaterial(
	int iter,
	int num_paths,
	ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	Geom* geoms,
	Triangle* tris,
	Material* materials,
	hipTextureObject_t* textureObjects,
	SegmentProcessingSettings settings
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths)
	{
		return;
	}

	PathSegment segment = pathSegments[idx];
	processSegment(segment, shadeableIntersections[idx], geoms, tris, materials, textureObjects, iter, idx, settings);

	pathSegments[idx] = segment;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];

#if DEBUG_NAN_MAGENTA
		if (isnan(iterationPath.color.x) || isnan(iterationPath.color.y) || isnan(iterationPath.color.z))
		{
			image[iterationPath.pixelIndex] = glm::vec3(1, 0, 1);
		}
		else
		{
			image[iterationPath.pixelIndex] += iterationPath.color;
		}
#else
	image[iterationPath.pixelIndex] += iterationPath.color;
#endif
	}
}

struct partition_predicate
{
	__host__ __device__ bool operator()(const PathSegment& ps)
	{
		return ps.remainingBounces > 0;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void Pathtracer::pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 512;

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_total_paths = dev_path_end - dev_paths;
	int num_valid_paths;

	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
	num_valid_paths = num_total_paths;

	while (num_valid_paths > 0) {
		// tracing
		dim3 numblocksPathSegmentTracing = (num_valid_paths + blockSize1d - 1) / blockSize1d;

#if FIRST_BOUNCE_CACHE
		if (guiData->firstBounceCache && !fbcNeedsRefresh && depth == 0)
		{
			thrust::copy(
				thrust::device,
				dev_thrust_intersections_fbc,
				dev_thrust_intersections_fbc + num_total_paths,
				dev_thrust_intersections
			);
		} 
		else
#endif
		{
			hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
				depth,
				num_valid_paths,
				dev_paths,
				dev_geoms,
				hst_scene->geoms.size(),
				dev_tris,
				dev_bvh_nodes,
				dev_bvh_tri_idx,
				dev_intersections
			);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
		}
		++depth;

		if (guiData->sortByMaterial)
		{
			thrust::sort_by_key(
				thrust::device,
				dev_thrust_intersections,
				dev_thrust_intersections + num_valid_paths,
				dev_thrust_paths
			);
			checkCUDAError("sort by material");
		}

		SegmentProcessingSettings settings;
		settings.russianRoulette = guiData->russianRoulette;

		shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
			iter,
			num_valid_paths,
			dev_intersections,
			dev_paths,
			dev_geoms,
			dev_tris,
			dev_materials,
			dev_textureObjects,
			settings
		);
		checkCUDAError("shade material");

#if FIRST_BOUNCE_CACHE
		if (guiData->firstBounceCache && depth == 1 && fbcNeedsRefresh)
		{
			thrust::copy(
				thrust::device,
				dev_thrust_intersections,
				dev_thrust_intersections + num_total_paths,
				dev_thrust_intersections_fbc
			);
			checkCUDAError("first bounce cache");
			fbcNeedsRefresh = false;
		}
#endif

		thrust::device_ptr<PathSegment> middle = thrust::partition(
			thrust::device,
			dev_thrust_paths,
			dev_thrust_paths + num_valid_paths,
			partition_predicate()
		);
		checkCUDAError("partition");

		num_valid_paths = middle - dev_thrust_paths;

		if (guiData != NULL)
		{
			guiData->tracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_total_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}

void Pathtracer::onCamChanged()
{
#if FIRST_BOUNCE_CACHE
	fbcNeedsRefresh = true;
#endif
}